// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

	
#include "hipblas.h"
	
int main(int argc, char **argv)
{
// 	int devID;
// 	hipDeviceProp_t props;
// 
// 	// This will pick the best possible CUDA capable device
// 	devID = findCudaDevice(argc, (const char **)argv);
// 
// 	//Get GPU information
// 	checkCudaErrors(hipGetDevice(&devID));
// 	checkCudaErrors(hipGetDeviceProperties(&props, devID));
// 	printf("Device %d: \"%s\" with Compute %d.%d capability\n",
// 		devID, props.name, props.major, props.minor);
	hipblasStatus_t status;
	status = cublasInit();
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
	 	printf("!!!! CUBLAS initialization error\n");
	 	return;
	}
	cublasShutdown();
	int imageSize = 512;
	unsigned int*p_devRgbU; 
	hipMalloc((void**)&p_devRgbU,/* sizeof(unsigned int)**/imageSize);
	hipFree(p_devRgbU);

	return 1;
}