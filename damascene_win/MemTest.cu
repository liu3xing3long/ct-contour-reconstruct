// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <fcntl.h>
#include <float.h>
#include <unistd.h>
#include "texton.h"
#include "convert.h"
#include "intervening.h"
#include "lanczos.h"
#include "stencilMVM.h"

#include "localcues.h"
#include "combine.h"
#include "nonmax.h"
#include "spectralPb.h"
#include "globalPb.h"
#include "skeleton.h"

#include "common_func.h"
#include "CircleTemplateTrace.h"
#include "LineSegTrace.h"

#include "pgm.cuh"

int main(int argc, char **argv)
{
	for (int i =0; i<= 74; i++)
	{
		int fileIdx = i;
		char filename[MAX_PATH];
		sprintf(filename, "%s%d", argv[1], fileIdx);
		char inputfile[MAX_PATH];
		char outputColorfile[MAX_PATH];

		char* period = strrchr(filename, '.');
		if (period == 0) {
			period = strrchr(filename, 0);
		}
		strncpy(inputfile, filename, period - filename);
		sprintf(&inputfile[0] + (period - filename) , "bin.pgm");

		strncpy(outputColorfile, filename, period - filename);
		sprintf(&outputColorfile[0] + (period - filename) , "_traced.pgm");

		/**null ��cutil�ڲ������ڴ�*/
		//float* data = NULL;	
		int width, height;

		//cutLoadPGMf(inputfile, (float**)&data, &width, &height);
		float* data = loadPGM(inputfile, &width, &height);
		assert(width > 0 && height > 0);

		CLineSegTrace trace;
		int nAmount = trace.initTracePoints((float*)data, width, height);
		printf("Outputting %s \n", inputfile);
		printf("Valid Pt number:%d \n", nAmount);
		trace.traceLineSegs();
		trace.debugPrintOutput(outputColorfile, 70, 100000);

/*		cutFree(data);*/
	}

	//system("pause");
	return 1;
}